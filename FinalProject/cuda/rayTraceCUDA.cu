#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include <stdio.h>
#include <iostream>
#include <hiprand.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include "cutil_math.h"

#define PI 3.14159265359f  
#define width 1024
#define height 1024
#define pixelSamples 4

//error handling micro, wrap it around function whenever possible
static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);
		system("pause");
		//exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

struct Ray{
	float3 o, d;
	__device__ 	Ray(float3 o_, float3 d_) :o(o_), d(d_){}
};
enum Refl_t { DIFF, SPEC, REFR };
struct Sphere
{
	float rad;
	float3 p, e, c;
	Refl_t refl;

	__device__ float intersect(const Ray &r) const { // returns distance, 0 if nohit
		float3 op = p - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0

		float t, eps = 1e-4, b = dot(op, r.d), det = b*b - dot(op, op) + rad*rad;
		if (det<0)
			return 0;
		else
			det = sqrtf(det);
		return (t = b - det)>eps ? t : ((t = b + det)>eps ? t : 0);
	}
};

//Scene Spheres 
__constant__ Sphere spheres[] = {
	{ 1e5f, { 1e5f + 1.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { 0.75f, 0.25f, 0.25f }, DIFF }, //Left 
	{ 1e5f, { -1e5f + 99.0f, 40.8f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .25f, .25f, .75f }, DIFF }, //Rght 
	{ 1e5f, { 50.0f, 40.8f, 1e5f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Back 
	{ 1e5f, { 50.0f, 40.8f, -1e5f + 600.0f }, { 0.0f, 0.0f, 0.0f }, { 1.00f, 1.00f, 1.00f }, DIFF }, //Frnt 
	{ 1e5f, { 50.0f, 1e5f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Botm 
	{ 1e5f, { 50.0f, -1e5f + 81.6f, 81.6f }, { 0.0f, 0.0f, 0.0f }, { .75f, .75f, .75f }, DIFF }, //Top 
	{ 16.5f, { 27.0f, 16.5f, 47.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // small sphere 1
	{ 16.5f, { 73.0f, 16.5f, 78.0f }, { 0.0f, 0.0f, 0.0f }, { 1.0f, 1.0f, 1.0f }, DIFF }, // small sphere 2
	{ 600.0f, { 50.0f, 681.6f - .77f, 81.6f }, { 2.0f, 1.8f, 1.6f }, { 0.0f, 0.0f, 0.0f }, DIFF }  // Light
};







inline float clamp(double x){ return x<0 ? 0 : x>1 ? 1 : x; }
inline int toInt(double x){ return int(pow(clamp(x), 1 / 2.2) * 255 + .5); }


__device__ inline bool intersect(const Ray&r, float&t, int&id)
{
	double n = sizeof(spheres) / sizeof(Sphere), d, inf = t = 1e20;
	for (int i = int(n); i--;) if ((d = spheres[i].intersect(r)) && d<t){ t = d; id = i; }	
	return t<inf;
}


__global__ void initialise_curand_on_kernels(hiprandState * state, unsigned long seed)
{
	//https://nidclip.wordpress.com/2014/04/02/cuda-random-number-generation/
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}
__device__ float generateRAND(hiprandState* globalState, int ind)
{
	//https://nidclip.wordpress.com/2014/04/02/cuda-random-number-generation/
	//copy state to local mem
	hiprandState localState = globalState[ind];
	//apply uniform distribution with calculated random
	float rndval = hiprand_uniform(&localState);
	//update state
	globalState[ind] = localState;
	//return value
	return rndval;
}

__device__ float3 Radiance(Ray &r, hiprandState* globalState, int ind)
{
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f);
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);

	for (int bounces = 0; bounces < 4; bounces++){
		float t;          
		int id = 0;

		if (!intersect(r, t, id))
			return make_float3(0.0f, 0.0f, 0.0f); 
		const Sphere &obj = spheres[id];
		float3 x = r.o + r.d*t;
		float3 n = normalize(x - obj.p);
		float3 nl = dot(n, r.d) < 0 ? n : n * -1;
		accucolor += mask * obj.e;

		float r1 = 2 * PI * generateRAND(globalState, ind);
		float r2 = generateRAND(globalState, ind);
		float r2s = sqrtf(r2);

		float3 w = nl;
		float3 u = normalize(cross((fabs(w.x) > .1 ? make_float3(0, 1, 0) : make_float3(1, 0, 0)), w));
		float3 v = cross(w, u);

		float3 d = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrtf(1 - r2));

		r.o = x + nl*0.05f;
		r.d = d;

		mask *= obj.c;
		mask *= dot(d, nl);
		mask *= 2;
	}

	return accucolor;
}
__global__ void pathTracing(float3*buffer_d, hiprandState* globalState)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = (height - y - 1)*width + x;

	

	Ray cam(make_float3(50, 52, 295.6), normalize(make_float3(0, -0.042612, -1)));
	float3 cx = make_float3(width * .5135 / height, 0.0f, 0.0f);
	float3 cy = normalize(cross(cx, cam.d)) * .5135;
	float3 r;
	r = make_float3(0.0f);
	for (int s = 0; s < pixelSamples; s++){
		float3 d = cam.d + cx*((.25 + x) / width - .5) + cy*((.25 + y) / height - .5);
		r = r + Radiance(Ray(cam.o + d * 40, normalize(d)), globalState, x)*(1. / pixelSamples);
	}
	buffer_d[i] = make_float3(clamp(r.x, 0.0f, 1.0f), clamp(r.y, 0.0f, 1.0f), clamp(r.z, 0.0f, 1.0f));
}

int main(){
	
	float3* buffer_d;

	HANDLE_ERROR(hipMalloc(&buffer_d, width*height*sizeof(float3)));

	dim3 threads(8, 8, 1);
	dim3 block(width / threads.x, height / threads.y, 1);

	//rand num
	srand(time(NULL));
	hiprandState* deviceStates;
	HANDLE_ERROR(hipMalloc(&deviceStates, 100 * sizeof(hiprandState)));
	initialise_curand_on_kernels << <100 / 1024 + 1, 1024 >> >(deviceStates, unsigned(time(NULL)));
	hipDeviceSynchronize();



	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, NULL));
	pathTracing << <block, threads >> >(buffer_d, deviceStates);
	hipDeviceSynchronize();
	HANDLE_ERROR(hipEventRecord(stop, NULL));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float time = 0.0f;
	HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));
	

	float3* buffer_h = new float3[width*height];
	HANDLE_ERROR(hipMemcpy(buffer_h, buffer_d, width*height*sizeof(float3), hipMemcpyDeviceToHost));

	hipFree(buffer_d);
	hipFree(deviceStates);

	FILE *f = fopen("image.ppm", "w");         // Write image to PPM file.
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i < width*height; i++){
		fprintf(f, "%d %d %d ", toInt(buffer_h[i].x), toInt(buffer_h[i].y), toInt(buffer_h[i].z));
	}
	fclose(f);

	printf("\nRendering time = %f (ms)", time);

	return 0;



}